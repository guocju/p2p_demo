#include "hip/hip_runtime.h"
#include "gpu_read_memory.cuh"
#include <cstdio>
#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}
    __global__ void readGPUMemory(int *d_data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > (N - 100) && idx < N) {
        printf("GPU Read: data[%d] = %d\n", idx, d_data[idx]);
    }
}


// 提供 C++ 兼容的封装函数
extern "C" void launchReadGPUMemory(int *gpu_ptr, int N) {
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    readGPUMemory<<<numBlocks, blockSize>>>(gpu_ptr, N);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    //hipDeviceSynchronize();  // 确保内核执行完成
}
